﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

// Simplified NVidia CUDA 11.7 Visual Studio Sample

void callCudaKernelWrapper(const int offset, const int *input, int* output, unsigned int size);

__global__ void offsetCounterKernel(const int offset, const int* input, int* output)
{
    int i = threadIdx.x;
    output[i] = input[i] + offset + i;
}

#define CheckCudaError(fn) if(fn != hipError_t::hipSuccess){ std::cout << #fn << "\n" << __FILE__ << ":" << __LINE__ << "  <-- FAILED!\n"; exit(0);}

int main()
{
    const   int arraySize         = 5;
    const   int offset            = 10;
    const   int input[arraySize]  = { 0, 0, 0, 0, 0 };
            int output[arraySize] = { 0 };

    // Add vectors in parallel.
    callCudaKernelWrapper(offset, input, output, arraySize);

    printf("{%d,%d,%d,%d,%d} + {%d} + {thread_index} = {%d,%d,%d,%d,%d}\n",
            input[0], input[1], input[2], input[3], input[4],
            offset,
            output[0], output[1], output[2], output[3], output[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    CheckCudaError(hipDeviceReset());

    return 0;
}

void callCudaKernelWrapper(const int offset, const int* input, int* output, unsigned int size)
{
    int * cuda_input = 0;
    int * cuda_output = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    CheckCudaError(hipSetDevice(0));

    // Allocate GPU buffers for three vectors (two input, one output)    .
    CheckCudaError(hipMalloc((void**)&cuda_input, size * sizeof(int)));
    CheckCudaError(hipMalloc((void**)&cuda_output, size * sizeof(int)));

    // Copy input vectors from host memory to GPU buffers.
    CheckCudaError(hipMemcpy(cuda_input, input, size * sizeof(int), hipMemcpyHostToDevice));

    // Launch a kernel on the GPU with one thread for each element.
    offsetCounterKernel <<<1, size>>>(offset, cuda_input, cuda_output);

    // Check for any errors launching the kernel
    CheckCudaError(hipGetLastError());
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    CheckCudaError(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    CheckCudaError(hipMemcpy(output, cuda_output, size * sizeof(int), hipMemcpyDeviceToHost));
}
